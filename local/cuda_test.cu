#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <chrono>

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__host__ __device__
void add(const float* a, const float* b, float* c, const int& i)
{
    c[i] = a[i] + b[i];
}

__global__
void kernel(const float* a, const float* b, float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    add(a, b, c, i);
}

int main()
{
    constexpr int N = 10;

    int block_count = N / 512 + 1;         // Optimal: power of 2
    int thread_count = N / block_count;    // Optimal: multiple of 32, range 128 and 512
    int batch_count = block_count * thread_count;

    printf("block_count: %i\nthread_count: %i\nbatch_count: %i\n", block_count, thread_count, batch_count);

    float a[N];
    float b[N];
    float c[N];

    for (int i = 0; i < N; ++i)
    {
        a[i] = 7;
        b[i] = 3;
    }

    float* cudaA;
    float* cudaB;
    float* cudaC;

    hipMalloc(&cudaA, N * sizeof(float));
    hipMalloc(&cudaB, N * sizeof(float));
    hipMalloc(&cudaC, N * sizeof(float));

    hipMemcpy(cudaA, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, N * sizeof(float), hipMemcpyHostToDevice);

    kernel << <block_count, thread_count >> > (cudaA, cudaB, cudaC);

    hipDeviceSynchronize();

    hipMemcpy(c, cudaC, N * sizeof(float), hipMemcpyDeviceToHost);

    bool equal = true;
    for (int i = 0; equal && i < N; ++i)
    {
        if (c[i] != 10)
            equal = false;
    }
    printf("%f (%i)\n", c[0], (int)equal);

    return 0;
}