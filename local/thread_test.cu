#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thread>
#include <iostream>
#include <chrono>

template <int N>
__host__ __device__ void work(const int* a, const int* b, int* c, int i)
{
    float j = 0.0f;
    for (int k = 0; k < 100000; ++k)
    {
        j += 0.1f;
    }

    c[i] = a[i] + b[i];
}

template <int N>
__global__ void kernel(const int* a, const int* b, int* c, const int per_thread)
{
    const int current = blockIdx.x * blockDim.x + threadIdx.x;
    const int begin = current * per_thread;
    const int end = begin + per_thread - 1;

    for (int i = begin; i <= end && i < N; ++i)
    {
        work<N>(a, b, c, i);
    }
}

void check(int* c, const int& N)
{
    bool equals = true;
    for (int i = 0; i < N && equals; ++i)
    {
        equals = (c[i] == 10);
    }

    printf("\tCorrect: %i (%i)\n", (int)equals, c[0]);
    memset(c, 0, N * sizeof(int));
}

template <int N>
void linear_test(const int* a, const int* b, int* c)
{
    auto linear_start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < N; ++i)
        work<N>(a, b, c, i);

    std::chrono::microseconds linear = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - linear_start);
    printf("Linear:   \t%lli", linear.count());
    check(c, N);
    printf("\t->%lli\n", linear.count() / N);
}

template <int N, int THREADS, int PER_THREAD>
void parallel_test(const int* a, const int* b, int* c)
{
    auto para_start = std::chrono::high_resolution_clock::now();

    std::thread threads[THREADS];

    for (int t = 0; t < THREADS; ++t)
    {
        threads[t] = std::thread([&, t]
            {
                const int begin = t * PER_THREAD;
                const int end = begin + PER_THREAD - 1;

                for (int i = begin; i <= end && i < N; ++i)
                {
                    work<N>(a, b, c, i);
                }
            });
    }

    for (int t = 0; t < THREADS; ++t)
        threads[t].join();

    std::chrono::microseconds para = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - para_start);
    printf("Parallel:\t%lli", para.count());
    check(c, N);
    printf("\t->%lli\n", para.count() / N);
}

template <int N>
void cuda_test(const int* a, const int* b, int* c, const int per_thread)
{
    auto cuda_start = std::chrono::high_resolution_clock::now();

    const int block_count = N / per_thread / 512 + 1;      // Optimal: power of 2
    const int thread_count = N / per_thread / block_count; // Optimal: multiple of 32, range 128 and 512
    const int used = per_thread * block_count * thread_count;

    printf("\nblock_count: %i thread_count: %i per_thread: %i unused: %i\n\n", block_count, thread_count, per_thread, N - used);

    int* cudaA;
    int* cudaB;
    int* cudaC;

    hipMalloc(&cudaA, used * sizeof(int));
    hipMalloc(&cudaB, used * sizeof(int));
    hipMalloc(&cudaC, used * sizeof(int));

    hipMemcpy(cudaA, a, used * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, used * sizeof(int), hipMemcpyHostToDevice);

    kernel<N> << <block_count, thread_count >> > (cudaA, cudaB, cudaC, 0);

    for (int i = used; i < N; ++i)
    {
        work<N>(a, b, c, i);
    }

    auto launched = std::chrono::high_resolution_clock::now();

    hipDeviceSynchronize();

    hipMemcpy(c, cudaC, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);

    std::chrono::microseconds cuda = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - cuda_start);
    std::chrono::microseconds cuda_launched = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - launched);

    printf("Cuda:   \t%llius (launched: %lldus)", cuda.count(), cuda_launched.count());
    check(c, N);
    printf("\t->%llius\n", cuda.count() / N);
}

int main()
{
    constexpr int N = 12000;
    constexpr int THREADS = 30;
    constexpr int PER_THREAD = N / THREADS;

    int* a = new int[N];
    int* b = new int[N];
    int* c = new int[N];

    for (int i = 0; i < N; ++i)
    {
        a[i] = 7;
        b[i] = 3;
    }

    linear_test<N>(a, b, c);
    parallel_test<N, THREADS, PER_THREAD>(a, b, c);
    cuda_test<N>(a, b, c, 1);
    auto start = std::chrono::high_resolution_clock::now();
    cuda_test<N>(a, b, c, 2);
    std::chrono::microseconds stop = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start);
    printf("%llu\n", stop.count());
    // cuda_test<N>(a, b, c, 3);
    // cuda_test<N>(a, b, c, 4);
    // cuda_test<N>(a, b, c, 8);
    // cuda_test<N>(a, b, c, 12);

    return 0;
}